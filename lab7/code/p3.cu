#include "hip/hip_runtime.h"
#include ""
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipDNN.h>
#include <iostream>

#define checkCUDNN(expression)                              \
{                                                           \
    hipdnnStatus_t status = (expression);                    \
    if (status != HIPDNN_STATUS_SUCCESS) {                   \
        std::cerr << "Error on line " << __LINE__ << ": "   \
            << hipdnnGetErrorString(status) << std::endl;    \
        std::exit(EXIT_FAILURE);                            \
    }                                                       \
}
 
int main(int argc, char * argv[] ) {

    // input: inputSize*inputSize*depth
    // kernel: kernelSize*kernelSize*depth
    // output: outputSize*outputSize

    int inputSize = 7;
    int depth = 3;
    int kernelSize = 3;
    int kernelNum = 3;
    int stride[3] = {1 , 2 , 3 };
    int pad[3] = {0,0,0};
    int outputSize[3];


    // 计算不同stride下需要的padding数量pad和output的规模outputSize

    for(int i = 0; i < kernelNum; i++) {
        if((inputSize - kernelSize)%stride[i] != 0) {
            pad[i] = (stride[i] - ((inputSize - kernelSize)%stride[i])) / 2;
        }
        outputSize[i] = (inputSize - kernelSize + 2*pad[i] ) / stride[i] + 1;
    }

    // ============================= 资源申请的初始化 =========================

    // ==== CPU资源申请和初始化
    // input:A kernel:kernel output:B

    float *A, *kernel[3], *B[3];
    A = (float *)malloc(sizeof(float)*inputSize*inputSize*depth);
    for(int i = 0; i < 3; i++) {
        kernel[i] = (float *)malloc(sizeof(float)*kernelSize*kernelSize*depth);
        B[i] = (float *)malloc(sizeof(float)*outputSize[i]*outputSize[i]*depth);
    }

   // 初始化input A
    for(int d = 0; d < depth; d++) {
        for(int i=0; i<inputSize*inputSize; i++) {
            A[d*inputSize*inputSize + i] = i;
        }
    }

    // 初始化kernel
    for(int i = 0; i < 3; i++){
        for(int j = 0; j < kernelSize*kernelSize*depth; j++) {
            kernel[i][j] = 1;
        }
    }

    // ==== GPU资源申请和初始化
    float *d_A, *d_kernel[3], *d_B[3];

    hipMalloc((void**)&d_A,sizeof(float)*inputSize*inputSize*depth);
    for(int i = 0; i < 3; i++) {
        hipMalloc((void**)&d_kernel[i], sizeof(float)*kernelSize*kernelSize*depth);
        hipMalloc((void**)&d_B[i],sizeof(float)*outputSize[i]*outputSize[i]*depth);
    }

    hipMemcpy(d_A,A,sizeof(float)*inputSize*inputSize*depth,hipMemcpyHostToDevice);
    for(int i = 0; i < 3; i++) {
        hipMemcpy(d_kernel[i],kernel[i],sizeof(float)*kernelSize*kernelSize*depth,hipMemcpyHostToDevice);
    }
    
    // ========================== cuDNN 调用 ===================
    hipdnnHandle_t cudnn[3];
    for( int i = 0; i < 3; i++ )
        checkCUDNN(hipdnnCreate(&cudnn[i]));

    hipdnnTensorDescriptor_t input_desc;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_desc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(
                input_desc,HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,
                1,depth,inputSize,inputSize));
    
    hipdnnFilterDescriptor_t filter_desc;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
    checkCUDNN(hipdnnSetFilter4dDescriptor(
                filter_desc,HIPDNN_DATA_FLOAT,HIPDNN_TENSOR_NCHW,
                1,depth,kernelSize,kernelSize));
    
    hipdnnConvolutionDescriptor_t conv_desc[3];
    for( int i = 0; i < 3; i++) {
        checkCUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc[i]));
        checkCUDNN(hipdnnSetConvolution2dDescriptor(
                    conv_desc[i],
                    pad[i],pad[i],stride[i],stride[i],1,1,
                    HIPDNN_CONVOLUTION,HIPDNN_DATA_FLOAT));
    }

    hipdnnTensorDescriptor_t output_desc[3];
    for( int i = 0; i < 3; i++ ) {
        checkCUDNN(hipdnnCreateTensorDescriptor(&output_desc[i]));
        checkCUDNN(hipdnnSetTensor4dDescriptor(
                    output_desc[i],HIPDNN_TENSOR_NCHW,HIPDNN_DATA_FLOAT,
                    1,1,outputSize[i],outputSize[i]));
    }
    
    hipdnnConvolutionFwdAlgo_t algo[3];
    size_t ws_size[3];
    float *ws_data[3];
    for( int i = 0; i < 3; i++ ) {
       hipdnnGetConvolutionForwardAlgorithm(
                    cudnn[i],
                    input_desc,filter_desc,conv_desc[i],output_desc[i],
                    HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo[i]);
    }
    
    for(int i = 0; i < 3; i++ )
    {
            checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(
                    cudnn[i],
                    input_desc,filter_desc,conv_desc[i],output_desc[i],
                    algo[i],&ws_size[i]));
        hipMalloc((void**)&ws_data[i],ws_size[i]);
    }
    
    //printf("1: %d \n 2: %d \n 3: %d \n", ws_size[0],ws_size[1],ws_size[2]);


    struct timeval start, end;
    gettimeofday( &start, NULL );

    float alpha = 1.0;
    float beta = 0.0;
    
    for(int i = 0; i < 3; i++ ) {
        checkCUDNN(hipdnnConvolutionForward(
                    cudnn[i],
                    &alpha,
                    input_desc,d_A,
                    filter_desc,d_kernel[i],
                    conv_desc[i],algo[i],ws_data[i],ws_size[i],
                    &beta,
                    output_desc[i],d_B[i]));
    }

    for( int i = 0; i < 3; i++ ) {
        hipMemcpy(B[i],d_B[i],sizeof(float)*outputSize[i]*outputSize[i]*depth,hipMemcpyDeviceToHost);
    }
    
    gettimeofday( &end, NULL );
    int timeuse = 1000000 * ( end.tv_sec - start.tv_sec ) + end.tv_usec - start.tv_usec;
    //printf("Block(%d,%d)   Grid(%d,%d).\n", Block.x, Block.y, Grid.x, Grid.y);
    printf("total time is %f ms\n", timeuse/(float)1000);

    FILE *b[3];
    b[0] = fopen("matrixB31.m", "wb");
    b[1] = fopen("matrixB32.m", "wb");
    b[2] = fopen("matrixB33.m", "wb");


    for(int k = 0; k < 3; k++ ) {
        fprintf(b[k], "B = [ \n");
        for (int i = 0; i < outputSize[k]; i++)
        {
            for (int j = 0; j < outputSize[k]; j++)
                fprintf(b[k], "%f ", B[k][i * outputSize[k] + j]);
            fprintf(b[k], "\n");
        }
        fprintf(b[k], "];");
    }

    // ============================= 资源释放 =========================

    free(A);
    hipFree(d_A);
    hipdnnDestroyTensorDescriptor(input_desc);
    hipdnnDestroyFilterDescriptor(filter_desc);

    for(int i = 0; i < 3; i++) {

        free(kernel[i]);
        free(B[i]);

        hipFree(d_B[i]);
        hipFree(d_kernel[i]);
        hipFree(ws_data[i]);

        hipdnnDestroyTensorDescriptor(output_desc[i]);
        hipdnnDestroyConvolutionDescriptor(conv_desc[i]);

    
        fclose(b[i]);
    }

    return 0;
}